#include "hip/hip_runtime.h"
#pragma once
#ifndef USE_RAND_DATA
#define USE_RAND_DATA
#endif

#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include <cstdlib>
#include <ctime>
#include <iostream>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include <stdio.h>

#include "helper.h"
#include "kernels.cu"
#include "helper_timer.h"

int main()
{
	StopWatchInterface
		*hTimer;
	clock_t 
		t_init, 
		t_end;
	Config 
		cfg;
	std::string 
		buf,
		fname;
	hipChannelFormatDesc
		formatdesc;
	float 
		*input_kernel;
	cv::Mat 
		img;
	int
		fft_bytes,
		kernel_bytes,
		data_bytes,
		kernel_dim,
		kernel_size,
		kernel_r,
		padding;
	Dim
		threadsByBlockDim,
		imgDim,
		fftDim;
	dim3
		threadBlock,
		BlockGrid;
	
	//initialize variables
	hTimer = NULL;
	sdkCreateTimer(&hTimer);

	threadsByBlockDim = Dim(16, 16);
	
#ifndef USE_RAND_DATA
	kernel_dim = 3;
	kernel_size = kernel_dim * kernel_dim;
	kernel_r = floor(kernel_dim);
	padding = kernel_dim - 1;

	//fname = "C:\\Users\\Public\\Pictures\\Sample Pictures\\Tulips.jpg";
	fname = "C:\\Users\\Public\\Pictures\\Sample Pictures\\Desert.jpg";
	input_kernel = new float[kernel_size] {
		-1, -2, -1,
			0, 0, 0,
			1, 2, 1
	};

	img = cv::imread(fname, CV_LOAD_IMAGE_GRAYSCALE);
	imgDim.cols = img.cols;
	imgDim.rows = img.rows;
	////
#else
	int _rows = 1000;
	int _cols = 1000;
	kernel_dim = 11;
	kernel_size = kernel_dim * kernel_dim;
	kernel_r = floor(kernel_dim);
	padding = kernel_dim - 1;

	input_kernel = new float[kernel_size];

	imgDim.cols = _rows;
	imgDim.rows = _cols;
#endif

	fftDim = setFFTdim(imgDim, padding);
	
	fft_bytes	 = SizeInBytes(fftDim, sizeof(Complex));
	kernel_bytes = SizeInBytes(kernel_size, sizeof(Complex));
	data_bytes	 = SizeInBytes(imgDim, sizeof(Complex));

	formatdesc = hipCreateChannelDesc<Complex>();
	hipfftPlan2d(&cfg.fftplan, fftDim.rows, fftDim.cols, HIPFFT_C2C);

	cfg.h_kernel = new Complex[kernel_bytes];
	cfg.h_data = new Complex[data_bytes];
	cfg.h_res_cpu = new Complex[data_bytes];
	cfg.h_res_gpu = new Complex[fft_bytes];
	hipMallocArray(&cfg.a_kernel, &formatdesc, kernel_dim, kernel_dim);
	hipMallocArray(&cfg.a_data, &formatdesc, imgDim.cols, imgDim.rows);
	hipMalloc((void **)&cfg.d_kernel, fft_bytes);
	hipMalloc((void **)&cfg.d_data, fft_bytes);

#ifndef USE_RAND_DATA
	for (size_t i = 0; i < kernel_size; i++)
		h_Set(cfg.h_kernel, i, 0, 1, input_kernel[i]);
	for (size_t i = 0; i < imgDim.rows; i++)
		for (size_t j = 0; j < imgDim.cols; j++)
			h_Set(cfg.h_data, i, j, imgDim.cols, img.at<uchar>(i, j));
#else
	float val = 1;
	for (size_t i = 0; i < kernel_size; i++)
	{
		val = 1.f;
		h_Set(cfg.h_kernel, i, 0, 1, val);
	}
		
	for (size_t i = 0; i < imgDim.rows; i++)
		for (size_t j = 0; j < imgDim.cols; j++)
		{
			val = (float)rand() / (float)RAND_MAX;
			h_Set(cfg.h_data, i, j, imgDim.cols, val);
		}
#endif

	//assigning memory at gpu
	hipMemset(cfg.d_kernel, 0, fft_bytes);
	hipMemset(cfg.d_data, 0, fft_bytes);
	hipMemcpyToArray(cfg.a_kernel, 0, 0, cfg.h_kernel, kernel_bytes, hipMemcpyHostToDevice);
	hipMemcpyToArray(cfg.a_data, 0, 0, cfg.h_data, data_bytes, hipMemcpyHostToDevice);
	hipBindTextureToArray(texKernel, cfg.a_kernel);
	hipBindTextureToArray(texData, cfg.a_data);

	//preprocessing data
	threadBlock = dim3(threadsByBlockDim.rows, threadsByBlockDim.cols);
	
	BlockGrid = dim3(Ceil(kernel_dim, threadBlock.x), Ceil(kernel_dim, threadBlock.y));
	padKernel << <BlockGrid, threadBlock >> >(
		cfg.d_kernel,
		fftDim,
		kernel_dim,
		kernel_r);

	BlockGrid = dim3(Ceil(fftDim.cols, threadBlock.x), Ceil(fftDim.rows, threadBlock.y));
	padData << <BlockGrid, threadBlock >> >(
		cfg.d_data,
		fftDim,
		imgDim,
		kernel_r);
	
	//convolution using convolution theorem
	std::cout << "Running with cufft.." << std::endl;
	sdkResetTimer(&hTimer);
	hipDeviceSynchronize();
	sdkStartTimer(&hTimer);
	hipfftExecC2C(cfg.fftplan, (hipfftComplex *)cfg.d_kernel, (hipfftComplex *)cfg.d_kernel, HIPFFT_FORWARD);
	hipfftExecC2C(cfg.fftplan, (hipfftComplex *)cfg.d_data, (hipfftComplex *)cfg.d_data, HIPFFT_FORWARD);
	product(cfg.d_data, cfg.d_kernel, fftDim);
	hipfftExecC2C(cfg.fftplan, (hipfftComplex *)cfg.d_data, (hipfftComplex *)cfg.d_data, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	sdkStopTimer(&hTimer);
	
	hipMemcpy(cfg.h_res_gpu, cfg.d_data, fft_bytes, hipMemcpyDeviceToHost);
	
	std::cout << "Running with cpu::conv2d.." << std::endl;
	t_init = clock();
	cpu_conv2d(cfg.h_res_cpu, cfg.h_data, cfg.h_kernel, imgDim, kernel_dim);
	t_end = clock();

	//saving
#ifndef RAND_DATA
	std::cout << "Saving results.." << std::endl;
	save("result.jpg", cfg.h_res_gpu, imgDim, fftDim);
#endif

	//comparing with conv2d
	std::cout << "Checking.." << std::endl;
	buf = check(cfg.h_res_cpu, cfg.h_res_gpu, imgDim.rows, imgDim.cols, fftDim.cols) ? ":)" : "-.-";
	std::cout << "..Test: " << buf.c_str() << std::endl;
	
	//release memory
	Shutdown(cfg);

	std::cout << "GPU: " << sdkGetTimerValue(&hTimer) << "ms" << std::endl;
	std::cout << "CPU: " << ((double)(t_end - t_init) / CLOCKS_PER_SEC) * 1e3 << "ms" << std::endl;
	
	//system("pause");
}
