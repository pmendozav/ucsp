#include "hip/hip_runtime.h"
#pragma once
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <type_traits>

#include "hip/hip_runtime_api.h"
#include <cmath>

#define norm1(x, y) (sqrt(x*x + y*y))
#define norm2(x, y) (fabs(x) + fabs(y))
#define norm3(x, y) (atan(y/x))

template<class T>
struct Mat
{
	T 
		*ptr;
	size_t 
		*pitch,
		*rows,
		*cols;

	inline __device__ T Get(size_t row, size_t col)
	{
		return *((T*)((char*)ptr + row * *pitch) + col);
	}

	inline __device__ T Set(size_t row, size_t col, T value)
	{
		*((T*)((char*)ptr + row * *pitch) + col) = value;
	}

	__device__ T eval(Mat<T> ref, size_t i_o, size_t j_o, size_t dh)
	{
		T result = 0;
		size_t i, j, i2, j2;
		size_t row_o, col_o;

		row_o = i_o - dh;
		col_o = j_o - dh;

		for (i = row_o, i2 = 0; i <= i_o + dh; i++, i2++)
		{
			for (j = col_o, j2 = 0; j <= j_o + dh; j++, j2++)
			{
				result += Get(i, j) * ref.Get(i2, j2);
			}
		}

		return result;
	}

	inline T h_Get(size_t row, size_t col)
	{
		return *((T*)((char*)ptr + row * *pitch) + col);
	}

	inline T h_Set(size_t row, size_t col, T value)
	{
		*((T*)((char*)ptr + row * *pitch) + col) = value;
	}

	T h_eval(Mat<T> ref, size_t i_o, size_t j_o, size_t dh)
	{
		T result = 0;
		size_t i, j, i2, j2;
		size_t row_o, col_o;

		row_o = i_o - dh;
		col_o = j_o - dh;

		for (i = row_o, i2 = 0; i <= i_o + dh; i++, i2++)
		{
			for (j = col_o, j2 = 0; j <= j_o + dh; j++, j2++)
			{
				result += h_Get(i, j) * ref.h_Get(i2, j2);
			}
		}

		return result;
	}
};

template<class T>
static inline Mat<T> createDevMat2d(size_t rows, size_t cols, T *src)
{
	Mat<T>
		dst;
	size_t
		pitch,
		width,
		height;

	width = cols * sizeof(T);
	height = rows;

	checkCudaErrors(hipMallocPitch(&(dst.ptr), &pitch, width, height));
	checkCudaErrors(hipMalloc(&(dst.pitch), sizeof(size_t)));
	checkCudaErrors(hipMalloc(&(dst.rows), sizeof(size_t)));
	checkCudaErrors(hipMalloc(&(dst.cols), sizeof(size_t)));

	checkCudaErrors(hipMemcpy2D(dst.ptr, pitch, src, width, width, height, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dst.pitch, &pitch, sizeof(size_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dst.rows, &rows, sizeof(size_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dst.cols, &cols, sizeof(size_t), hipMemcpyHostToDevice));

	return dst;
}

template<class T>
static inline Mat<T> createHostMat2d(size_t rows, size_t cols, T *src)
{
	Mat<T>
		dst;
	size_t
		pitch,
		width,
		height;

	width = cols * sizeof(T);
	height = rows;

	dst.ptr = src;
	dst.pitch = new size_t[1]{width};
	dst.rows = new size_t[1]{rows};
	dst.cols = new size_t[1]{cols};

	return dst;
}

template<class T>
__global__ void conv2d2(Mat<T> img, Mat<T> kernel, T *result)
{
	size_t dh;
	size_t rows, cols;
	size_t tid_x, tid_y, offset_x, offset_y;

	rows = *(img.rows);
	cols = *(img.cols);
	dh = floor(static_cast<T>(*(kernel.rows) / 2));

	offset_x = gridDim.x * blockDim.x;
	offset_y = gridDim.y * blockDim.y;

	for (tid_x = threadIdx.x + blockIdx.x * blockDim.x; tid_x < rows; tid_x += offset_x)
		for (tid_y = threadIdx.y + blockIdx.y * blockDim.y; tid_y < cols; tid_y += offset_y)
			result[tid_x * cols + tid_y] = img.eval(kernel, tid_x, tid_y, dh);
}

template<class T>
static void cpu_conv2d2(Mat<T> img, Mat<T> kernel, T *result)
{
	size_t dh;
	size_t rows, cols;
	size_t tid_x, tid_y;

	rows = *(img.rows);
	cols = *(img.cols);
	dh = floor(static_cast<T>(*(kernel.rows) / 2));

	for (tid_x = dh; tid_x < rows - dh; tid_x++)
		for (tid_y = dh; tid_y < cols - dh; tid_y++)
			result[tid_x * cols + tid_y] = img.h_eval(kernel, tid_x, tid_y, dh);
}

template<class T>
__global__ void sobelFilter(Mat<T> img, Mat<T> Gx, Mat<T> Gy, T *result)
{
	size_t dh;
	size_t rows, cols;
	size_t tid_x, tid_y, offset_x, offset_y;
	T x, y;

	rows = *(img.rows);
	cols = *(img.cols);
	dh = floor(static_cast<T>(*(Gx.rows) / 2));

	offset_x = gridDim.x * blockDim.x;
	offset_y = gridDim.y * blockDim.y;

	for (tid_x = threadIdx.x + blockIdx.x * blockDim.x; tid_x < rows; tid_x += offset_x)
		for (tid_y = threadIdx.y + blockIdx.y * blockDim.y; tid_y < cols; tid_y += offset_y)
		{
			x = img.eval(Gx, tid_x, tid_y, dh);
			y = img.eval(Gy, tid_x, tid_y, dh);
			result[tid_x * cols + tid_y] = static_cast<T>(norm2(x, y));
		}
}

template<class T>
static void cpu_sobelFilter(Mat<T> img, Mat<T> Gx, Mat<T> Gy, T *result)
{
	size_t dh;
	size_t rows, cols;
	size_t tid_x, tid_y;
	T x, y;

	rows = *(img.rows);
	cols = *(img.cols);
	dh = floor(static_cast<T>(*(Gx.rows) / 2));

	for (tid_x = dh; tid_x < rows - dh; tid_x++)
		for (tid_y = dh; tid_y < cols - dh; tid_y++)
		{
			x = img.h_eval(Gx, tid_x, tid_y, dh);
			y = img.h_eval(Gy, tid_x, tid_y, dh);
			result[tid_x * cols + tid_y] = static_cast<T>(norm2(x, y));
		}
}

template<class T>
static T *Mat2Pointer(cv::Mat img)
{
	T *ptr = new T[img.rows * img.cols];
	for (int i = 0; i < img.rows; i++)
		memcpy(&(ptr[i*img.cols]), img.ptr<T>(i, 0), img.cols * sizeof(T));
	return ptr;
}

template<class T>
static cv::Mat Pointer2Mat(T *ptr, size_t rows, size_t cols)
{
	cv::Mat img;

	if (std::is_same<T, float>::value)
		img = cv::Mat::zeros(rows, cols, CV_32FC1);
	else
		img = cv::Mat::zeros(rows, cols, CV_8UC1);

	for (int i = 0; i < img.rows; i++)
		memcpy(img.ptr<T>(i, 0), &(ptr[i*img.cols]), img.cols * sizeof(T));
	return img;
}

typedef float type_ref;

template __global__ void conv2d2<type_ref>(Mat<type_ref> img, Mat<type_ref> kernel, type_ref *result);
template __global__ void sobelFilter<type_ref>(Mat<type_ref> img, Mat<type_ref> Gx, Mat<type_ref> Gy, type_ref *result);

static void test_gpu()
{
	//hipEvent_t start, stop;
	cv::Mat
		in;
	size_t
		rows,
		cols;
	type_ref
		*src_img,
		*src_kernel_g_x,
		*src_kernel_g_y;
	Mat<type_ref>
		d_img,
		d_kernel_g_x,
		d_kernel_g_y;
	type_ref
		*d_result,
		*h_result;
	dim3
		Blocks,
		Threads;
	float
		t_gx = 0;

	//hipEventCreate(&start);
	//hipEventCreate(&stop);

	Blocks = dim3(4, 4);
	Threads = dim3(5, 5);

	in = cv::imread("modelos.bmp", CV_LOAD_IMAGE_GRAYSCALE);
	in.convertTo(in, CV_32FC1);

	rows = in.rows;
	cols = in.cols;
	src_img = Mat2Pointer<type_ref>(in);

	d_img = createDevMat2d(rows, cols, src_img);

	h_result = new type_ref[rows * cols];
	checkCudaErrors(hipMalloc((void **)&d_result, rows * cols * sizeof(type_ref)));

	//Gx
	printf("Testing Gx-filter..\n");
	src_kernel_g_x = new type_ref[3 * 3]{
		-1, 0, 1,
			-2, 0, 2,
			-1, 0, 1};

	d_kernel_g_x = createDevMat2d(3, 3, src_kernel_g_x);
	//hipEventRecord(start);
	conv2d2<type_ref> << <Blocks, Threads >> >(d_img, d_kernel_g_x, d_result);
	//checkCudaErrors(hipDeviceSynchronize());
	//hipEventRecord(stop);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&t_gx, start, stop);
	checkCudaErrors(hipMemcpy(h_result, d_result, rows * cols * sizeof(type_ref), hipMemcpyDeviceToHost));

	in = Pointer2Mat<type_ref>(h_result, in.rows, in.cols);
	in.convertTo(in, CV_8UC1);
	imwrite("Gx.tif", in);

	//Gy
	printf("Testing Gy-filter..\n");
	src_kernel_g_y = new type_ref[3 * 3]{
		1, 2, 1,
			0, 0, 0,
			-1, -2, -1};

	d_kernel_g_y = createDevMat2d(3, 3, src_kernel_g_y);
	conv2d2<type_ref> << <Blocks, Threads >> >(d_img, d_kernel_g_y, d_result);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy(h_result, d_result, rows * cols * sizeof(type_ref), hipMemcpyDeviceToHost));

	in = Pointer2Mat<type_ref>(h_result, in.rows, in.cols);
	in.convertTo(in, CV_8UC1);
	imwrite("Gy.tif", in);

	//Sobel Filter
	printf("Testing gradient..\n");
	sobelFilter<type_ref> << <Blocks, Threads >> >(d_img, d_kernel_g_x, d_kernel_g_y, d_result);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy(h_result, d_result, rows * cols * sizeof(type_ref), hipMemcpyDeviceToHost));

	in = Pointer2Mat<type_ref>(h_result, in.rows, in.cols);
	in.convertTo(in, CV_8UC1);
	imwrite("SobelFilter.tif", in);
}

static void test_cpu()
{
	cv::Mat
		in;
	size_t
		rows,
		cols;
	type_ref
		*src_img,
		*src_kernel_g_x,
		*src_kernel_g_y;
	Mat<type_ref>
		h_img,
		kernel_g_x,
		kernel_g_y;
	type_ref
		*d_result,
		*h_result;
	dim3
		Blocks,
		Threads(5, 5);

	Blocks = dim3(4, 4);
	Threads = dim3(5, 5);

	in = cv::imread("modelos.bmp", CV_LOAD_IMAGE_GRAYSCALE);
	in.convertTo(in, CV_32FC1);

	rows = in.rows;
	cols = in.cols;
	src_img = Mat2Pointer<type_ref>(in);

	h_img = createHostMat2d(rows, cols, src_img);

	h_result = new type_ref[rows * cols];

	src_kernel_g_x = new type_ref[3 * 3]{
		-1, 0, 1,
			-2, 0, 2,
			-1, 0, 1};
	src_kernel_g_y = new type_ref[3 * 3]{
		1, 2, 1,
			0, 0, 0,
			-1, -2, -1};

	//Gx
	printf("Testing Gx-filter..\n");
	kernel_g_x = createHostMat2d(3, 3, src_kernel_g_x);
	cpu_conv2d2<type_ref>(h_img, kernel_g_x, h_result);
	
	in = Pointer2Mat<type_ref>(h_result, in.rows, in.cols);
	in.convertTo(in, CV_8UC1);
	imwrite("Gx_cpu.tif", in);
	//return;
	//Gy
	printf("Testing Gy-filter..\n");
	kernel_g_y = createHostMat2d(3, 3, src_kernel_g_y);
	cpu_conv2d2<type_ref>(h_img, kernel_g_y, h_result);

	in = Pointer2Mat<type_ref>(h_result, in.rows, in.cols);
	in.convertTo(in, CV_8UC1);
	imwrite("Gy_cpu.tif", in);

	//Sobel Filter
	printf("Testing gradient..\n");
	cpu_sobelFilter<type_ref>(h_img, kernel_g_x, kernel_g_y, h_result);

	in = Pointer2Mat<type_ref>(h_result, in.rows, in.cols);
	in.convertTo(in, CV_8UC1);
	imwrite("SobelFilter_cpu.tif", in);
}


int main()
{
	printf("******Testing cpu..\n");
	test_cpu();
	printf("******Testing gpu..\n");
	test_gpu();

	return 0;
}
