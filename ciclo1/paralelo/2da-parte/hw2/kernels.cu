#include "hip/hip_runtime.h"
#pragma once

__device__ void Set(Complex *v, const size_t &i, const  size_t &j, const size_t &offset, float2 &val)
{
	v[__mul24(i, offset) + j] = val;
}
__device__ void Mult(Complex& a, const Complex &b, const float &c = 1)
{
	Complex t = { c * (a.x * b.x - a.y * b.y), c * (a.y * b.x + a.x * b.y) };
	a = t;
}
__global__ void padKernel(
	Complex *d_PaddedKernel,
	const Dim dim,
	const int kernel_dim,
	const int kernel_r)
{
	const int j = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
	const int i = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
	int _i, _j;

	if (j < kernel_dim && i < kernel_dim)
	{
		_j = j - kernel_r; if (_j < 0) _j += dim.cols;
		_i = i - kernel_r; if (_i < 0) _i += dim.rows;
		Set(d_PaddedKernel, _i, _j, dim.cols, tex2D(texKernel, (float)j + 0.5f, (float)i + 0.5f));
	}
}
__global__ void padData(
	Complex *d_PaddedData,
	const Dim dim,
	const Dim dim_ref,
	const int kernel_r)
{
	const int j = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
	const int i = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
	const int borderW = dim_ref.cols + kernel_r;
	const int borderH = dim_ref.rows + kernel_r;
	int _i, _j;

	if (j < dim.cols && i < dim.rows)
	{
		if (j < dim_ref.cols) _i = j;
		if (i < dim_ref.rows) _j = i;
		if (j >= dim_ref.cols && j < borderW) _i = dim_ref.cols - 1;
		if (i >= dim_ref.rows && i < borderH) _j = dim_ref.rows - 1;
		if (j >= borderW) _i = 0;
		if (i >= borderH) _j = 0;

		Set(d_PaddedData, i, j, dim.cols, tex2D(texData, (float)_i + 0.5f, (float)_j + 0.5f));
	}
}
__global__ void _product(
	Complex *d_PaddedData,
	const Complex *d_PaddedKernel,
	const int N)
{
	const int tid = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
	const int offset = __mul24(blockDim.x, gridDim.x);
	const float alpha = 1.0f / (float)N;

	for (size_t i = tid; i < N; i += offset)
		Mult(d_PaddedData[i], d_PaddedKernel[i], alpha);
}

void product(Complex *A, Complex *B, Dim dim)
{
	_product << <8, 1024 >> >(A, B, dim.cols * dim.rows);
}